#include "hip/hip_runtime.h"
/*
libMeshlessVis
Copyright (C) 2008 Andrew Corrigan

This program is free software; you can redistribute it and/or
modify it under the terms of the GNU General Public License
as published by the Free Software Foundation; either version 2
of the License, or (at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
*/

#include "meshless_vis.h"

#include <GL/glew.h>
#include <hipfft/hipfft.h>
#include <cutil.h>
#include <cuda_gl_interop.h>
#include <stdio.h>
#include <math.h>

#include "fourier_transform.h"



__device__ float dot(float3 a, float3 b)
{
	return a.x*b.x + a.y*b.y + a.z*b.z;
}

__device__ bool point_in_box(float2 point, float4 bounding_box)
{
	return (point.x >= bounding_box.x && point.y >= bounding_box.y && point.x <= bounding_box.z && point.y <= bounding_box.w);
}

__global__ void mask_fully_aliased_terms(int number_of_terms, MeshlessDataset meshless_dataset, bool* mask, float4 bounding_box, float3 u_axis, float3 v_axis)
{
	int index = (blockDim.x*blockIdx.x + threadIdx.x);
	if(index > number_of_terms) return;
	
	float3 center = meshless_dataset.d_constraints[index].position;
	float radius = meshless_dataset.d_radii[index];
	
	float2 corner1 = make_float2(dot(center, u_axis)+radius,dot(center, v_axis)+radius);
	float2 corner2 = make_float2(corner1.x, corner1.y-2*radius);
	float2 corner3 = make_float2(corner1.x-2*radius, corner1.y);
	float2 corner4 = make_float2(corner3.x, corner2.y);

	mask[index] = (point_in_box(corner1, bounding_box) && point_in_box(corner2, bounding_box) && point_in_box(corner3, bounding_box) && point_in_box(corner4, bounding_box));
}


void cull_fully_aliased_points(MeshlessDataset* meshless_dataset, VisConfig* vis_config)
{
	if(vis_config->cull_fully_aliased_terms)
	{
		dim3 block_size(vis_config->_block_length);
		int number_of_terms = get_number_of_terms(meshless_dataset);
		dim3 grid((number_of_terms/block_size.x)+(number_of_terms%block_size.x > 0)?1:0);
		//float px = 1.0f/vis_config->step_size.x, py = 1.0f/vis_config->step_size.y;
		float4 bounding_box = make_float4(0.0f, 0.0f, 1.0f/vis_config->step_size.x, 1.0f/vis_config->step_size.y);
		bool* mask;
	
		CUDA_SAFE_CALL(hipMalloc((void**)&mask, sizeof(bool)*number_of_terms));
				
		mask_fully_aliased_terms<<<grid,block_size>>>(number_of_terms, *meshless_dataset, mask, bounding_box, vis_config->u_axis, vis_config->v_axis);

		/*
		Constraint* culled_constraints;
		float* culled_radii;
		unsigned int number_not_culled;
		CUDPPScanConfig* config;
		cudppInitializeScan(config);
		cudppCompact((void*)culled_constraints, mask, meshless_data->d_constraints, number_of_terms, config, &number_not_culled);
		cudppCompact((void*)culled_radii, mask, meshless_data->d_radii, number_of_terms, config, &number_not_culled);
		cudppFinalizeScan(config);
		*/
		
		CUDA_SAFE_CALL(hipFree(mask));
	}
}

VisConfig* vis_config_create(bool automatic_d_image, float2 step_size, int2 cutoff_frequency, float3 u_axis, float3 v_axis, int2 number_of_samples, int fvr_block_length, int minimum_number_of_blocks)
{
	VisConfig* vis_config = (VisConfig*)malloc(sizeof(VisConfig));
	vis_config->step_size = step_size;
	vis_config->u_axis = u_axis;
	vis_config->v_axis = v_axis;
	vis_config->cutoff_frequency = cutoff_frequency;
	vis_config->cull_fully_aliased_terms = false;
	vis_config_change_fvr_block_length(vis_config, fvr_block_length);
	vis_config->_d_image = 0;	//need this otherwise it will try to delete whatever random location d_image currently points to
	vis_config_change_number_of_samples(vis_config, automatic_d_image, number_of_samples);
	vis_config->minimum_number_of_blocks = minimum_number_of_blocks;
	
	
	hipDeviceProp_t prop;
	int dev;
	
	CUDA_SAFE_CALL(hipGetDevice(&dev));
	CUDA_SAFE_CALL(hipGetDeviceProperties(&prop, dev));
	
	printf("Name:                    %s\n", prop.name);
	printf("Global Memory:           %d\n", prop.totalGlobalMem);
	printf("Shared Memory per Block: %d\n", prop.sharedMemPerBlock);
	printf("Register per Block:      %d\n", prop.regsPerBlock);
	printf("Warp Size:               %d\n", prop.warpSize);
	printf("Memory Pitch:            %d\n", prop.memPitch);
	printf("Max Threads Per Block:   %d\n", prop.maxThreadsPerBlock);
	printf("Max Threads Dimension:   %d %d %d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
	printf("Max Grid Size:           %d %d %d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
	printf("Constant Memory:         %d\n", prop.totalConstMem);
	printf("Version:                 %d.%d\n", prop.major, prop.minor);
	printf("Clock Rate:              %d\n", prop.clockRate);
	printf("Texture Alignment:       %d\n", prop.textureAlignment);
	
	if(prop.major==1 && prop.minor < 1)
	{
		printf("Warning: Your card does not have compute capability 1.1\n");
	}
	
	if(prop.maxThreadsPerBlock != prop.maxThreadsDim[0]) printf("libMeshlessVis WARNING: Assumption made that maxThreadsPerBlock == maxThreadsDim[0]\n");
	vis_config->_block_length = prop.maxThreadsDim[0];
	
	return vis_config;
}

VisConfig* vis_config_get_default()
{
	float2 step_size = make_float2(0.1f, 0.1f);
	int2 cutoff_frequency = make_int2(128,128);
	float3 u_axis = make_float3(1.0f, 0.0f, 0.0f);
	float3 v_axis = make_float3(0.0f, 1.0f, 0.0f);
	int2 number_of_samples = make_int2(512,512);
	int fvr_block_length = 256;
	int minimum_number_of_blocks = 0;
	VisConfig* vis_config = vis_config_create(true, step_size, cutoff_frequency, u_axis, v_axis, number_of_samples, fvr_block_length, minimum_number_of_blocks);
	return vis_config;
}

bool vis_config_check(VisConfig* vis_config)
{
	hipDeviceProp_t prop;
	int dev;
	CUDA_SAFE_CALL(hipGetDevice(&dev));
	CUDA_SAFE_CALL(hipGetDeviceProperties(&prop, dev));
	
	if(vis_config->_fvr_block_length < 64) return false;	/* Minimum specified by Appendix A.1 of the NVIDIA CUDA 1.1 Programming Guide */
	if(vis_config->_fvr_block_length < prop.warpSize) return false;	/* We want a fully populated warp */
	if(vis_config->_block_length < 64) return false;	/* Minimum specified by Appendix A.1 of the NVIDIA CUDA 1.1 Programming Guide */
	if(vis_config->_block_length < prop.warpSize) return false;	/* We want a fully populated warp */

	if(vis_config->_fvr_block_length > prop.maxThreadsPerBlock) return false;
	if(vis_config->_fvr_block_length > prop.maxThreadsDim[0]) return false;
	if((2*vis_config->cutoff_frequency.x*vis_config->cutoff_frequency.y)%vis_config->_fvr_block_length) return false;
		
	if(vis_config->_block_length > prop.maxThreadsPerBlock) return false;
	if(vis_config->_block_length > prop.maxThreadsDim[0]) return false;	
	if((vis_config->_number_of_samples.x*vis_config->_number_of_samples.y)%vis_config->_block_length) return false;

	return true;
}

void vis_config_change_fvr_block_length(VisConfig* vis_config, int fvr_block_length)
{
	vis_config->_fvr_block_length = fvr_block_length;
}

void vis_config_change_number_of_samples(VisConfig* vis_config, bool automatic_d_image, int2 number_of_samples)
{
	vis_config_clean(vis_config);
	vis_config->_number_of_samples = number_of_samples;
	vis_config->_automatic_d_image = automatic_d_image;
	if(automatic_d_image) CUDA_SAFE_CALL(hipMalloc((void**)&vis_config->_d_image, sizeof(float2)*vis_config->_number_of_samples.x*vis_config->_number_of_samples.y));
	CUDA_SAFE_CALL(hipMalloc((void**)&vis_config->_d_freq_image, sizeof(float2)*vis_config->_number_of_samples.x*vis_config->_number_of_samples.y));
	// Previously freq_image was not a power of two size in order to minimize memory usage, so a thread block size of 1x1 was used when scaling the image
	// Forsaking the reduced memory cost, by storing the power of two sized image, more optimal thread block sizes can be in the zero and scale kernels, improving performance in certain cases
	hipfftPlan2d(&vis_config->_plan, vis_config->_number_of_samples.x, vis_config->_number_of_samples.y, HIPFFT_C2R);
}

void vis_config_manual_d_image(VisConfig* vis_config, float2* d_image)
{
	vis_config->_d_image = d_image;
}

void vis_config_clean(VisConfig* vis_config)
{
	if(vis_config->_d_image != 0)
	{
		if(vis_config->_automatic_d_image) CUDA_SAFE_CALL(hipFree(vis_config->_d_image));
		CUDA_SAFE_CALL(hipFree(vis_config->_d_freq_image));
		hipfftDestroy(vis_config->_plan);		
	}
}

void vis_config_destroy(VisConfig* vis_config)
{
	vis_config_clean(vis_config);
	free(vis_config);
}

void vis_config_compute_scale(VisConfig* vis_config)
{
	// scale the unnormalized FFT by our discrete step size
	vis_config->_scale = vis_config->step_size.x * vis_config->step_size.y;
}

template <typename T>
T* load_into_device(T* h_data, int N)
{
	T* d_data;
	if(h_data == 0)
	{
		d_data = 0;
	}
	else
	{
		CUDA_SAFE_CALL(hipMalloc((void**)&d_data, sizeof(T)*N));
		CUDA_SAFE_CALL(hipMemcpy(d_data, h_data, sizeof(T)*N, hipMemcpyHostToDevice));
	}
	return d_data;
}

void vis_register_meshless_dataset(MeshlessDataset* meshless_dataset)
{
	int number_of_terms = get_number_of_terms(meshless_dataset);
	meshless_dataset->d_constraints = load_into_device(meshless_dataset->h_constraints, number_of_terms);
	meshless_dataset->d_radii = load_into_device(meshless_dataset->h_radii, number_of_terms);
}

void vis_unregister_meshless_dataset(MeshlessDataset* meshless_dataset)
{
	CUDA_SAFE_CALL(hipFree(meshless_dataset->d_constraints));
	CUDA_SAFE_CALL(hipFree(meshless_dataset->d_radii));
}

void vis_fourier_volume_rendering(MeshlessDataset meshless_dataset, VisConfig* vis_config)
{

	// Initialize the image's Fourier transform to zero since a lowpass filter will be used
	hipMemset((void*)vis_config->_d_freq_image, 0, sizeof(float2)*vis_config->_number_of_samples.x*vis_config->_number_of_samples.y);

	// Sample the image's Fourier transform
	fourier_transform(meshless_dataset, vis_config);

	// Compute, using a Complex-To-Real FFT, the inverse Fourier transform of the Fourier transform of the image. 	
	hipfftExecC2R(vis_config->_plan, (hipfftComplex*)vis_config->_d_freq_image, (hipfftReal*)vis_config->_d_image);
}

void vis_copy_to_host(VisConfig* vis_config, float* h_image)
{
	CUDA_SAFE_CALL(hipMemcpy(h_image, vis_config->_d_image, sizeof(float)*vis_config->_number_of_samples.x*vis_config->_number_of_samples.y, hipMemcpyDeviceToHost));
}

void vis_register_buffer_object(GLuint buffer_object) { CUDA_SAFE_CALL(cudaGLRegisterBufferObject(buffer_object)); }
void vis_unregister_buffer_object(GLuint buffer_object) { CUDA_SAFE_CALL(cudaGLUnregisterBufferObject(buffer_object)); }
void vis_opengl_fourier_volume_rendering(MeshlessDataset meshless_dataset, VisConfig* vis_config, GLuint registered_buffer_object)
{
	CUDA_SAFE_CALL(cudaGLMapBufferObject( (void**)&vis_config->_d_image, registered_buffer_object));
	vis_fourier_volume_rendering(meshless_dataset,vis_config);	
	CUDA_SAFE_CALL(cudaGLUnmapBufferObject(registered_buffer_object));
}
