#include "hip/hip_runtime.h"
/*
libMeshlessVis
Copyright (C) 2008 Andrew Corrigan

This program is free software; you can redistribute it and/or
modify it under the terms of the GNU General Public License
as published by the Free Software Foundation; either version 2
of the License, or (at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
*/

#include "meshless_vis.h"

#include <GL/glew.h>
#include <hipfft/hipfft.h>
#include <cutil.h>
#include <cudpp/cudpp.h>
#include <cuda_gl_interop.h>
#include <stdio.h>
#include <math.h>

#include "fourier_transform.h"



__device__ __host__ float dot(float3 a, float3 b)
{
	return a.x*b.x + a.y*b.y + a.z*b.z;
}

__device__ __host__ unsigned int point_in_box(float2 point, float4 bounding_box)
{
	return (point.x >= bounding_box.x && point.y >= bounding_box.y && point.x <= bounding_box.z && point.y <= bounding_box.w);
}

__global__ void compute_mask(Group group, unsigned int* mask, float4 bounding_box, float3 u_axis, float3 v_axis)
{
	int index = (blockDim.x*blockIdx.x + threadIdx.x);
	float3 center = group.d_constraints[index].position;
	float radius = group.d_radii[index];
	
	float2 icenter = make_float2(dot(center, u_axis),dot(center, v_axis));

	float2 corner1 = make_float2(icenter.x+radius, icenter.y+radius);
	float2 corner2 = make_float2(icenter.x+radius, icenter.y-radius);
	float2 corner3 = make_float2(icenter.x-radius, icenter.y-radius);
	float2 corner4 = make_float2(icenter.x-radius, icenter.y+radius);	
	
	mask[index] = index < group.number_of_terms && (point_in_box(corner1, bounding_box) && point_in_box(corner2, bounding_box) && point_in_box(corner3, bounding_box) && point_in_box(corner4, bounding_box));
}

template<typename T>
__global__ void compact(T* in, T* out, unsigned int* mask, unsigned int* indices)
{
	int index = (blockDim.x*blockIdx.x + threadIdx.x);
	if(mask[index]) out[index] = in[indices[index]];
}

void cull_fully_aliased_terms(MeshlessDataset* meshless_dataset, VisConfig* vis_config)
{
	int max_number_of_terms = meshless_dataset->groups[0].d_number_of_terms;
	for(int k = 1; k < meshless_dataset->number_of_groups; k++) { if(max_number_of_terms < meshless_dataset->groups[k].number_of_terms) max_number_of_terms = meshless_dataset->groups[k].d_number_of_terms; }
	
	unsigned int* mask, *indices;
	CUDA_SAFE_CALL(hipMalloc((void**)&mask, sizeof(unsigned int)*max_number_of_terms));
	CUDA_SAFE_CALL(hipMalloc((void**)&indices, sizeof(unsigned int)*max_number_of_terms));

	unsigned int* cpu_mask1 = (unsigned int*)malloc(sizeof(unsigned int)*max_number_of_terms);
	unsigned int* cpu_mask2 = (unsigned int*)malloc(sizeof(unsigned int)*max_number_of_terms);

	Constraint* compacted_constraints; float* compacted_radii;
	CUDA_SAFE_CALL(hipMalloc((void**)&compacted_constraints, sizeof(Constraint)*max_number_of_terms));
	CUDA_SAFE_CALL(hipMalloc((void**)&compacted_radii, sizeof(float)*max_number_of_terms));

	CUDPPScanConfig config;
    config.direction = CUDPP_SCAN_FORWARD;
    config.exclusivity = CUDPP_SCAN_EXCLUSIVE;
    config.maxNumElements = max_number_of_terms;
    config.maxNumRows = 1;
    config.datatype = CUDPP_UINT;
    config.op = CUDPP_ADD;
    
    cudppInitializeScan(&config);

	dim3 block_length(vis_config->block_length);
	float w = 1.0f/vis_config->step_size.x, h = 1.0f/vis_config->step_size.y;
	float4 bounding_box = make_float4(0, 0, w/2, h/2);
	for(int k = 0; k < meshless_dataset->number_of_groups; k++)
	{
		Group group = meshless_dataset->groups[k];

		// Step 1: Compute the mask
		dim3 grid(group.d_number_of_terms / vis_config->block_length); if(group.d_number_of_terms%vis_config->block_length > 0) printf("error\n");
		compute_mask<<<grid, block_length>>>(group, mask, bounding_box, vis_config->u_axis, vis_config->v_axis);
		
		// Step 2: Compute the indices
		cudppScan(indices, mask, group.number_of_terms, &config);
		
		// Step 3: Map data
		CUDA_SAFE_CALL(hipMemset(compacted_radii, 0, group.d_number_of_terms*sizeof(float)));
		compact<<<grid, block_length>>>(group.d_radii, compacted_radii, mask, indices);
		CUDA_SAFE_CALL(hipMemcpy(group.d_constraints, compacted_constraints, group.d_number_of_terms*sizeof(float), hipMemcpyDeviceToDevice));

		CUDA_SAFE_CALL(hipMemset(compacted_constraints, 0, group.d_number_of_terms*sizeof(Constraint)));
		compact<<<grid, block_length>>>(group.d_constraints, compacted_constraints, mask, indices);
		CUDA_SAFE_CALL(hipMemcpy(group.d_constraints, compacted_constraints, group.d_number_of_terms*sizeof(Constraint), hipMemcpyDeviceToDevice));
	}

	cudppFinalizeScan(&config);
	
	free(cpu_mask1);
	free(cpu_mask2);
		
	CUDA_SAFE_CALL(hipFree(mask));
	CUDA_SAFE_CALL(hipFree(indices));
	CUDA_SAFE_CALL(hipFree(compacted_constraints));
	CUDA_SAFE_CALL(hipFree(compacted_radii));

}

VisConfig* vis_config_create(bool automatic_d_image, float2 step_size, int2 cutoff_frequency, float3 u_axis, float3 v_axis, int2 number_of_samples, int block_length, int number_of_partial_sums)
{
	VisConfig* vis_config = (VisConfig*)malloc(sizeof(VisConfig));
	vis_config->step_size = step_size;
	vis_config->u_axis = u_axis;
	vis_config->v_axis = v_axis;
	vis_config->cull_fully_aliased_terms = false;
	vis_config->_number_of_samples = number_of_samples;
	vis_config->_cutoff_frequency = cutoff_frequency;
	vis_config->block_length = block_length;
	vis_config->_number_of_partial_sums = number_of_partial_sums;

	CUDA_SAFE_CALL(hipMalloc((void**)&vis_config->_d_freq_image, sizeof(float2)*2*vis_config->_cutoff_frequency.x*vis_config->_cutoff_frequency.y*vis_config->_number_of_partial_sums));
	CUDA_SAFE_CALL(hipMalloc((void**)&vis_config->_d_freq_image_arranged, sizeof(float2)*vis_config->_number_of_samples.x*(vis_config->_number_of_samples.y/2+1)));

	vis_config->_automatic_d_image = automatic_d_image;
	if(vis_config->_automatic_d_image) CUDA_SAFE_CALL(hipMalloc((void**)&vis_config->_d_image, sizeof(float)*vis_config->_number_of_samples.x*vis_config->_number_of_samples.y));
	CUFFT_SAFE_CALL(hipfftPlan2d(&vis_config->_plan, vis_config->_number_of_samples.x, vis_config->_number_of_samples.y, HIPFFT_C2R));
	
	hipDeviceProp_t prop;
	int dev;
	
	CUDA_SAFE_CALL(hipGetDevice(&dev));
	CUDA_SAFE_CALL(hipGetDeviceProperties(&prop, dev));
	
	printf("Name:                    %s\n", prop.name);
	printf("Global Memory:           %d\n", prop.totalGlobalMem);
	printf("Shared Memory per Block: %d\n", prop.sharedMemPerBlock);
	printf("Register per Block:      %d\n", prop.regsPerBlock);
	printf("Warp Size:               %d\n", prop.warpSize);
	printf("Memory Pitch:            %d\n", prop.memPitch);
	printf("Max Threads Per Block:   %d\n", prop.maxThreadsPerBlock);
	printf("Max Threads Dimension:   %d %d %d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
	printf("Max Grid Size:           %d %d %d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
	printf("Constant Memory:         %d\n", prop.totalConstMem);
	printf("Version:                 %d.%d\n", prop.major, prop.minor);
	printf("Clock Rate:              %d\n", prop.clockRate);
	printf("Texture Alignment:       %d\n", prop.textureAlignment);
	
	if(prop.major==1 && prop.minor < 1)
	{
		printf("Warning: Your card does not have compute capability 1.1\n");
	}
	
	if(prop.maxThreadsPerBlock != prop.maxThreadsDim[0]) printf("libMeshlessVis WARNING: Assumption made that maxThreadsPerBlock == maxThreadsDim[0]\n");
	
	return vis_config;
}

VisConfig* vis_config_get_default()
{
	float2 step_size = make_float2(0.1f, 0.1f);
	int2 cutoff_frequency = make_int2(128,128);
	float3 u_axis = make_float3(1.0f, 0.0f, 0.0f);
	float3 v_axis = make_float3(0.0f, 1.0f, 0.0f);
	int2 number_of_samples = make_int2(512,512);
	int block_length = 256;
	int number_of_partial_sums = 1;
	VisConfig* vis_config = vis_config_create(true, step_size, cutoff_frequency, u_axis, v_axis, number_of_samples, block_length, number_of_partial_sums);
	return vis_config;
}

bool vis_config_check(VisConfig* vis_config)
{
	hipDeviceProp_t prop;
	int dev;
	CUDA_SAFE_CALL(hipGetDevice(&dev));
	CUDA_SAFE_CALL(hipGetDeviceProperties(&prop, dev));

	if(vis_config->block_length < 64) return false;	/* Minimum specified by Appendix A.1 of the NVIDIA CUDA 1.1 Programming Guide */
	if(vis_config->block_length < prop.warpSize) return false;	/* We want a fully populated warp */
	if(vis_config->block_length > prop.maxThreadsPerBlock) return false;
	if(vis_config->block_length > prop.maxThreadsDim[0]) return false;
	if((2*vis_config->_cutoff_frequency.x*vis_config->_cutoff_frequency.y)%vis_config->block_length) return false;
	
	if(2*vis_config->_cutoff_frequency.x > vis_config->_number_of_samples.x) return false;
	if(2*vis_config->_cutoff_frequency.y > vis_config->_number_of_samples.y) return false;
	
	if(vis_config->_number_of_partial_sums < 1) return false;

	return true;
}

void vis_config_change_number_of_samples(VisConfig* vis_config, int2 number_of_samples)
{
	vis_config->_number_of_samples = number_of_samples;

	if(vis_config->_automatic_d_image) 
	{
		CUDA_SAFE_CALL(hipFree(vis_config->_d_image));
		CUDA_SAFE_CALL(hipMalloc((void**)&vis_config->_d_image, sizeof(float)*vis_config->_number_of_samples.x*vis_config->_number_of_samples.y));
	}

	CUDA_SAFE_CALL(hipFree(vis_config->_d_freq_image_arranged));
	CUDA_SAFE_CALL(hipMalloc((void**)&vis_config->_d_freq_image_arranged, sizeof(float2)*vis_config->_number_of_samples.x*(vis_config->_number_of_samples.y/2+1)));
	
	CUFFT_SAFE_CALL(hipfftDestroy(vis_config->_plan));
	CUFFT_SAFE_CALL(hipfftPlan2d(&vis_config->_plan, vis_config->_number_of_samples.x, vis_config->_number_of_samples.y, HIPFFT_C2R));
}

void vis_config_change_cutoff_frequency(VisConfig* vis_config, int2 cutoff_frequency)
{
	vis_config->_cutoff_frequency = cutoff_frequency;
	CUDA_SAFE_CALL(hipFree(vis_config->_d_freq_image));
	CUDA_SAFE_CALL(hipMalloc((void**)&vis_config->_d_freq_image, sizeof(float2)*2*vis_config->_cutoff_frequency.x*vis_config->_cutoff_frequency.y*vis_config->_number_of_partial_sums));
}

void vis_config_change_number_of_partial_sums(VisConfig* vis_config, int number_of_partial_sums)
{
	vis_config->_number_of_partial_sums = number_of_partial_sums;

	CUDA_SAFE_CALL(hipFree(vis_config->_d_freq_image));
	CUDA_SAFE_CALL(hipMalloc((void**)&vis_config->_d_freq_image, sizeof(float2)*2*vis_config->_cutoff_frequency.x*vis_config->_cutoff_frequency.y*vis_config->_number_of_partial_sums));
}

void vis_config_manual_d_image(VisConfig* vis_config, float* d_image)
{
	vis_config->_d_image = d_image;
}

void vis_config_destroy(VisConfig* vis_config)
{
	if(vis_config->_automatic_d_image) CUDA_SAFE_CALL(hipFree(vis_config->_d_image));
	CUDA_SAFE_CALL(hipFree(vis_config->_d_freq_image_arranged));
	CUDA_SAFE_CALL(hipFree(vis_config->_d_freq_image));
	CUFFT_SAFE_CALL(hipfftDestroy(vis_config->_plan));
	free(vis_config);
}

void vis_config_compute_scale(VisConfig* vis_config)
{
	// scale the unnormalized FFT by our discrete step size
	vis_config->_scale = vis_config->step_size.x * vis_config->step_size.y;
}

template <typename T>
T* load_into_device(T* h_data, int N, int integer_multiple_of, int& rounded_N)
{
	T* d_data;
	if(h_data == 0)
	{
		d_data = 0;
	}
	else
	{
		int k = (N / integer_multiple_of) + min(1, N%integer_multiple_of);
		rounded_N = integer_multiple_of*k;
		CUDA_SAFE_CALL(hipMalloc((void**)&d_data, sizeof(T)*rounded_N));
		CUDA_SAFE_CALL(hipMemset((void*)d_data, 0, sizeof(T)*rounded_N));
		CUDA_SAFE_CALL(hipMemcpy(d_data, h_data, sizeof(T)*N, hipMemcpyHostToDevice));
	}
	return d_data;
}

void vis_register_meshless_dataset(VisConfig* vis_config, MeshlessDataset* meshless_dataset)
{
	for(int j = 0; j != meshless_dataset->number_of_groups; j++)
	{
		meshless_dataset->groups[j].d_constraints = load_into_device(meshless_dataset->groups[j].h_constraints, meshless_dataset->groups[j].number_of_terms, vis_config->_number_of_partial_sums*vis_config->block_length, meshless_dataset->groups[j].d_number_of_terms);
		meshless_dataset->groups[j].d_radii = load_into_device(meshless_dataset->groups[j].h_radii, meshless_dataset->groups[j].number_of_terms, vis_config->_number_of_partial_sums*vis_config->block_length, meshless_dataset->groups[j].d_number_of_terms);
	}
}

void vis_unregister_meshless_dataset(VisConfig* vis_config, MeshlessDataset* meshless_dataset)
{
	for(int j = 0; j != meshless_dataset->number_of_groups; j++)
	{
		CUDA_SAFE_CALL(hipFree(meshless_dataset->groups[j].d_constraints));
		CUDA_SAFE_CALL(hipFree(meshless_dataset->groups[j].d_radii));
	}
}

__global__ void reduce_partial_sums(VisConfig vis_config)
{
	// TODO: replace this kernel with a call to cudppMultiScan (if possible)
	
	int index = (blockDim.x*blockIdx.x + threadIdx.x);
	int size = 2*vis_config._cutoff_frequency.x*vis_config._cutoff_frequency.y;
	
	float2 partial_sum = vis_config._d_freq_image[index];
	float2 term;
	for(int i = 1; i < vis_config._number_of_partial_sums; i++)
	{
		term = vis_config._d_freq_image[index+i*size];
		partial_sum = make_float2(partial_sum.x+term.x, partial_sum.y+term.y);
	}
	vis_config._d_freq_image[index] = partial_sum;
}

__global__ void arrange_samples(VisConfig vis_config)
{
	
	int index = (blockDim.x*blockIdx.x + threadIdx.x);
	int x = index % (2*vis_config._cutoff_frequency.x);
	if(x == vis_config._cutoff_frequency.x) return;
	int y = index / (2*vis_config._cutoff_frequency.x);
	int index_x = x;
	if(x > vis_config._cutoff_frequency.x) 
	{
		x = x-(2*vis_config._cutoff_frequency.x);
		index_x = (x+vis_config._number_of_samples.x);
	}
	vis_config._d_freq_image_arranged[index_x*(vis_config._number_of_samples.y/2+1)+y] = vis_config._d_freq_image[index];
}

void vis_fourier_volume_rendering(MeshlessDataset* meshless_dataset, VisConfig* vis_config)
{
//	cull_fully_aliased_terms(meshless_dataset, vis_config);
	
	CUDA_SAFE_CALL(hipMemset((void*)vis_config->_d_freq_image_arranged, 0, sizeof(float2)*vis_config->_number_of_samples.x*(vis_config->_number_of_samples.y/2+1)));
	fourier_transform(meshless_dataset, vis_config); CUT_CHECK_ERROR("fourier_transform failed");

	dim3 block_size(vis_config->block_length);
	dim3 cutoff_grid(2*vis_config->_cutoff_frequency.x*vis_config->_cutoff_frequency.y / vis_config->block_length);	
	if(vis_config->_number_of_partial_sums > 1)
	{
		reduce_partial_sums<<<cutoff_grid, block_size>>>(*vis_config); CUT_CHECK_ERROR("reduce_partial_sums failed");
	}
	arrange_samples<<<cutoff_grid, block_size>>>(*vis_config); CUT_CHECK_ERROR("arrange_samples failed");

	CUFFT_SAFE_CALL(hipfftExecC2R(vis_config->_plan, (hipfftComplex*)vis_config->_d_freq_image_arranged, (hipfftReal*)vis_config->_d_image));
}

void vis_copy_to_host(VisConfig* vis_config, float* h_image)
{
	CUDA_SAFE_CALL(hipMemcpy(h_image, vis_config->_d_image, sizeof(float)*vis_config->_number_of_samples.x*vis_config->_number_of_samples.y, hipMemcpyDeviceToHost));
}

void vis_opengl_fourier_volume_rendering(MeshlessDataset* meshless_dataset, VisConfig* vis_config, GLuint buffer_object)
{
	CUDA_SAFE_CALL(cudaGLRegisterBufferObject(buffer_object));
	CUDA_SAFE_CALL(cudaGLMapBufferObject( (void**)&vis_config->_d_image, buffer_object));
	vis_fourier_volume_rendering(meshless_dataset,vis_config);	
	CUDA_SAFE_CALL(cudaGLUnmapBufferObject(buffer_object));
	CUDA_SAFE_CALL(cudaGLUnregisterBufferObject(buffer_object));
}
