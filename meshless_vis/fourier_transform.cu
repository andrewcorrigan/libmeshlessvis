#include "hip/hip_runtime.h"
/*

libMeshlessVis
Copyright (C) 2008 Andrew Corrigan

This program is free software; you can redistribute it and/or
modify it under the terms of the GNU General Public License
as published by the Free Software Foundation; either version 2
of the License, or (at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.

*/

#include "fourier_transform.h"
#include <string.h>
#include <math_functions.h>
#include <hip/hip_math_constants.h>

#ifndef CUDART_2PI_F
#define CUDART_2PI_F 6.283185307179586476925286766559f
#endif

inline __device__ float fourier_transform_sph(float r)
{
	if(r >= 0.06f) 
	{
		float m = HIP_PI_F*r;
		float cos_2_m, sin_2_m;
		__sincosf(2.0f*m, &sin_2_m, &cos_2_m);	// this makes a huge difference
		return ((2.3561944901923448f)/(m*m*m*m*m*m))*(cos_2_m-1.0f)*(cos_2_m+m*sin_2_m-1.0f);		// computing m**6 this way is much faster than pow(m,6)
	}
	return (HIP_PI_F + r*(-0.007968913156311f + r*(-18.293608272337678f)));		// a quadratic interpolant, note that HIP_PI_F is the value at zero
}

inline __device__ float fourier_transform_gaussian(float r)
{
	return __expf(HIP_PI_F*r*r);
}

inline __device__ float fourier_transform_wendland_d3_c2(float r)
{
	if(r >= 0.24f) 
	{
		float m = HIP_PI_F*r;
		float cos_2_m, sin_2_m;
		__sincosf(2.0f*m, &sin_2_m, &cos_2_m);	// this makes a huge difference
		float m_2 = m*m;
		float m_4 = m_2*m_2;
		return ((HIP_PI_F*7.5f)/(m_4*m_4))*(4.0f*m_2 - 6.0f + (6.0f-m_2)*cos_2_m+4.5f*m*sin_2_m);
	}
	return 0.299199300341890f + r*(-0.002379178586124f + r*(-0.370530218163545f));
}

inline __device__ float dot(float3 a, float3 b)
{
	return a.x*b.x + a.y*b.y + a.z*b.z;
}

template <int block_length, bool is_first_group, BasisFunctionId basis_function_id, bool has_radii>
__global__ void sample_fourier_transform_over_grid(/*int d_number_of_terms, float* d_radii, Constraint* d_constraints,*/Group group, VisConfig vis_config)
{
	extern __shared__ float shared[];
	Constraint* ds_constraints = (Constraint*)shared;
	float* ds_radii;
	if(has_radii) ds_radii = (float*)(ds_constraints + block_length);

	int image_size = 2*vis_config._cutoff_frequency.x*vis_config._cutoff_frequency.y;
	int index = (block_length*blockIdx.x + threadIdx.x);
	int x = index % (2*vis_config._cutoff_frequency.x);
	if(x > vis_config._cutoff_frequency.x) x = x-(2*vis_config._cutoff_frequency.x);
	int y = (index % image_size) / (2*vis_config._cutoff_frequency.x);
	int partial_sum_index = index / image_size;
	int number_of_terms_per_partial_sum = group.d_number_of_terms / vis_config._number_of_partial_sums;
	int first_term = partial_sum_index*number_of_terms_per_partial_sum;
	int last_term = first_term + number_of_terms_per_partial_sum;

	// compute the image space coordinates
	float fu = vis_config.step_size.x*x, fv = vis_config.step_size.y*y;
	
	// map from image space into frequency space
	float3 f_coord = make_float3(fu*vis_config.u_axis.x + fv*vis_config.v_axis.x, fu*vis_config.u_axis.y + fv*vis_config.v_axis.y, fu*vis_config.u_axis.z + fv*vis_config.v_axis.z);

	// we don't use the fast sqrt since it's only computed once
	float r = sqrtf((float)(fu*fu + fv*fv));

	// begin the loop
	float2 sum = make_float2(0.0f, 0.0f);
	float sin_v, cos_v;
	Constraint constraint;
	float r0;
	int thread_index = threadIdx.x;
	for(unsigned int k = first_term; k < last_term; k += block_length) 
	{
		// step 1: stage global memory into shared memory.  this access is coalesced, and should be minimal since there is one 128-bit read and one 32-bit read
		ds_constraints[thread_index] = group.d_constraints[k + thread_index];
		if(has_radii) ds_radii[thread_index] = group.d_radii[k + thread_index];
		__syncthreads();
		
		//step 2: for each point, accumulate the terms in shared memory (there are no bank conflicts here, the same data is broadcast to every thread)		
		for(unsigned int j = 0; j != block_length; j++)
		{
			constraint = ds_constraints[j];
			float term;
			if(has_radii)
			{
				r0 = ds_radii[j];
				if     (basis_function_id == SPH)            term = fourier_transform_sph(r*r0);
				else if(basis_function_id == GAUSSIAN)       term = fourier_transform_gaussian(r*r0);
				else if(basis_function_id == WENDLAND_D3_C2) term = fourier_transform_wendland_d3_c2(r*r0);
			}
			else
			{
				if     (basis_function_id == SPH)            term = fourier_transform_sph(r);
				else if(basis_function_id == GAUSSIAN)       term = fourier_transform_gaussian(r);
				else if(basis_function_id == WENDLAND_D3_C2) term = fourier_transform_wendland_d3_c2(r);
			}
			__sincosf(CUDART_2PI_F*dot(f_coord, constraint.position), &sin_v, &cos_v);	// using this __sincos2f function compared to sin and cos makes a huge difference
			sum.x += constraint.weight*term*cos_v;
			sum.y += constraint.weight*term*sin_v;
		}
		__syncthreads();
	}

	if(is_first_group)
	{
		vis_config._d_freq_image[index] = make_float2(sum.x*vis_config._scale, -sum.y*vis_config._scale);
	}
	else
	{
		float2 prev = vis_config._d_freq_image[index];
		vis_config._d_freq_image[index] = make_float2(prev.x + sum.x*vis_config._scale, prev.y - sum.y*vis_config._scale);
	}
}

template <int block_length, bool is_first_group, BasisFunctionId basis_function_id>
void fourier_transform_level_2(Group* group, VisConfig* vis_config)
{
	dim3 block_size(vis_config->block_length);
	dim3 cutoff_grid((2*vis_config->_cutoff_frequency.x*vis_config->_cutoff_frequency.y*vis_config->_number_of_partial_sums) / vis_config->block_length);	

	if(group->d_radii) sample_fourier_transform_over_grid <block_length, is_first_group, basis_function_id, true>  <<<cutoff_grid, block_size, vis_config->block_length*sizeof(float)*5>>> (*group, *vis_config);
	else               sample_fourier_transform_over_grid <block_length, is_first_group, basis_function_id, false> <<<cutoff_grid, block_size, vis_config->block_length*sizeof(float)*4>>> (*group, *vis_config);
}

template <int block_length, bool is_first_group>
void fourier_transform_level_1(Group* group, VisConfig* vis_config)
{
	if     (group->basis_function_id == SPH)            fourier_transform_level_2 <block_length, is_first_group, SPH>            (group, vis_config);
	else if(group->basis_function_id == GAUSSIAN)       fourier_transform_level_2 <block_length, is_first_group, GAUSSIAN>       (group, vis_config);
	else if(group->basis_function_id == WENDLAND_D3_C2) fourier_transform_level_2 <block_length, is_first_group, WENDLAND_D3_C2> (group, vis_config);
}

void fourier_transform(MeshlessDataset* meshless_dataset, VisConfig* vis_config)
{
	if (meshless_dataset->number_of_groups < 1) return;

	vis_config_compute_scale(vis_config);

	if     (vis_config->block_length == 512) fourier_transform_level_1 <512, true> (meshless_dataset->groups, vis_config);
	else if(vis_config->block_length == 256) fourier_transform_level_1 <256, true> (meshless_dataset->groups, vis_config);
	else if(vis_config->block_length == 128) fourier_transform_level_1 <128, true> (meshless_dataset->groups, vis_config);
	else if(vis_config->block_length ==  64) fourier_transform_level_1 < 64, true> (meshless_dataset->groups, vis_config);
	else if(vis_config->block_length ==  32) fourier_transform_level_1 < 32, true> (meshless_dataset->groups, vis_config);

	for(int i = 1; i < meshless_dataset->number_of_groups; i++)
	{
		if     (vis_config->block_length == 512) fourier_transform_level_1 <512, false> (meshless_dataset->groups+i, vis_config);
		else if(vis_config->block_length == 256) fourier_transform_level_1 <256, false> (meshless_dataset->groups+i, vis_config);
		else if(vis_config->block_length == 128) fourier_transform_level_1 <128, false> (meshless_dataset->groups+i, vis_config);
		else if(vis_config->block_length ==  64) fourier_transform_level_1 < 64, false> (meshless_dataset->groups+i, vis_config);
		else if(vis_config->block_length ==  32) fourier_transform_level_1 < 32, false> (meshless_dataset->groups+i, vis_config);
	}
}
